
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <functional>
#include <chrono>
#include <unistd.h>

#ifndef BUF_KIND
#define BUF_KIND 0
#endif

#if(BUF_KIND == 0)
#define BUF_TYPE float
#else
#define BUF_TYPE double
#endif

#define CudaWrap(EXP) \
{ \
    auto ret = EXP; \
    if (ret != hipSuccess) { \
        std::cerr << "Error! " << hipGetErrorString(ret) << " (" << ret << ")" << std::endl; \
        return 1; \
    }\
}

__global__
void addKernel(const BUF_TYPE* A, BUF_TYPE* B, size_t size) {
    size_t i = blockIdx.x*blockDim.x+threadIdx.x;
    if (i < size) {
        B[i] = A[i] + B[i];
    }
}

int main() {
    // Initialize random number generator
    std::mt19937_64 generator;
    generator.seed(42);
    std::uniform_real_distribution<BUF_TYPE> distribution(-1., 1.);
    auto gen = std::bind(distribution, generator);

    // Initialize arrays
    size_t num_gen = (1<<(30))/sizeof(BUF_TYPE);

    // Allocate arrays.
    BUF_TYPE* array1 = new BUF_TYPE[num_gen];
    BUF_TYPE* array2 = new BUF_TYPE[num_gen];

    // Allocate Device arrays.
    BUF_TYPE* dev_1 = nullptr;
    BUF_TYPE* dev_2 = nullptr;
    CudaWrap(hipMalloc(&dev_1, num_gen*sizeof(BUF_TYPE)));
    CudaWrap(hipMalloc(&dev_2, num_gen*sizeof(BUF_TYPE)));

    // Fill arrays with values
    for(size_t i=0; i < num_gen; ++i) {
        array1[i] = gen();
        array2[i] = gen();
    }

    // Copy data to Device
    hipMemcpy(dev_1, array1, num_gen*sizeof(BUF_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(dev_2, array2, num_gen*sizeof(BUF_TYPE), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    auto start = std::chrono::high_resolution_clock::now();
    // Compute on Device
    int blockSize = 256;
    int numBlocks = (num_gen+blockSize-1)/(blockSize);
    addKernel<<<numBlocks,blockSize>>>(array1, array2, num_gen);
    // Copy data out of Device
    hipMemcpy(array2, dev_2, num_gen*sizeof(BUF_TYPE), hipMemcpyDeviceToHost);
    // Wait for operations to finish.
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();


    // Do something with the arrays so the addition isn't optimized out.
    BUF_TYPE sum = 0.;
    for(size_t i=0; i < num_gen; ++i) {
        sum += array2[i];
    }

    std::cout << sum << std::endl;
    std::cout << std::hexfloat;
    std::cout << sum << std::endl;

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);

    std::cout << "Took " << duration.count() << " milliseconds" << std::endl;

    hipFree(dev_1);
    hipFree(dev_2);

    delete [] array1;
    delete [] array2;

    return 0;
}
